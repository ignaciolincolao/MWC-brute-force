#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <cstdlib>
#include <random>
#include <cstdint>
#include <cstring>
#include <chrono>
#include <vector>


#include <Dataset.cuh>
#include <Coalition.cuh>

using namespace std;


int main(int argc, char* argv[]){
    if (argc < 3) {
        std::cout << "Faltan argumentos. Uso: programa <nombre> <puntos> <seed>\n";
        return 1;
    }

    std::string nombre = argv[1];
    int puntos = std::stoi(argv[2]);
    std::string mseed = argv[3];

    cout << mseed << endl;
    Dataset DATOS(nombre);
    //Dataset DATOS("data/test/points_100.txt");
    //Dataset DATOS("data/test/points_40.txt");
    //Dataset DATOS("data/votes.json","JSON");
    DATOS.printXY();
    cout << DATOS.getDistanciaHost(0,1) << " - " << DATOS.getDistanciaDevice(0,1) << endl;
    int quorum = trunc(DATOS.X.size() / 2)+1;

    Coalition COALITION(quorum,DATOS.X.size(),DATOS.distMatrix_device,1024,1024); 
    //Coalition COALITION(quorum,DATOS.X.size(),DATOS.distMatrix_device,5078,1024); 
    // Time variable initialization for execution calculation
    auto initial_time = chrono::high_resolution_clock::now();
    cout << "Generando combinatoria!" << endl;
    COALITION.BestSolution();

    auto final_time = chrono::high_resolution_clock::now();
    double time_taken = chrono::duration_cast<chrono::nanoseconds>(final_time - initial_time).count();



    time_taken *= 1e-9;
    cout << "Time:"<< fixed << time_taken << setprecision(9) << endl;
    cout << "Minimum Fitness:" << COALITION.bestFitness << endl;
    cout << "Coalition:" << endl;
    for (size_t i = 0; i < quorum; i++)
    {
        cout << COALITION.bestSolution[i] << ",";
    }


    ofstream result_file;
    result_file.open("../data/result/"+std::to_string(puntos)+"_seed_"+mseed+".txt");
    result_file << "Time:"<< fixed << time_taken << setprecision(9) << endl;
    result_file << "Minimum Fitness:" << COALITION.bestFitness << endl;
    result_file << "N_BLOCK: " << COALITION.nBlock << " N_THREADS:" << COALITION.nThread << endl;
    result_file << "Coalition:" << endl;
    for (size_t i = 0; i < quorum; i++)
    {
        result_file << COALITION.bestSolution[i] << ",";
    }
    result_file.close();  


    return 0;
}