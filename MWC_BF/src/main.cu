#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <cstdlib>
#include <random>
#include <cstdint>
#include <cstring>
#include <chrono>
#include <vector>


#include <Dataset.cuh>
#include <Coalition.cuh>

using namespace std;


int main(int argc, char* argv[]){
    if (argc < 5) {
        std::cout << "Faltan argumentos. Uso: programa <nombre> <puntos> <puntos_izq> <puntos_der>\n";
        return 1;
    }

    std::string nombre = argv[1];
    int puntos = std::stoi(argv[2]);
    int puntos_izq = std::stoi(argv[3]);
    int puntos_der = std::stoi(argv[4]);
    std::string mseed = argv[5];

    cout << mseed << endl;
    Dataset DATOS(nombre);
    //Dataset DATOS("data/test/points_100.txt");
    //Dataset DATOS("data/test/points_40.txt");
    //Dataset DATOS("data/votes.json","JSON");
    DATOS.printXY();
    cout << DATOS.getDistanciaHost(0,1) << " - " << DATOS.getDistanciaDevice(0,1) << endl;
    int quorum = trunc(DATOS.X.size() / 2)+1;

    Coalition COALITION(quorum,DATOS.X.size(),DATOS.distMatrix_device,1024,1024); 
    //Coalition COALITION(quorum,DATOS.X.size(),DATOS.distMatrix_device,5078,1024); 
    // Time variable initialization for execution calculation
    auto initial_time = chrono::high_resolution_clock::now();
    cout << "Generando combinatoria!" << endl;
    COALITION.BestSolution();

    auto final_time = chrono::high_resolution_clock::now();
    double time_taken = chrono::duration_cast<chrono::nanoseconds>(final_time - initial_time).count();



    time_taken *= 1e-9;
    cout << "Time:"<< fixed << time_taken << setprecision(9) << endl;
    cout << "Minimum Fitness:" << COALITION.bestFitness << endl;
    cout << "Coalition:" << endl;
    for (size_t i = 0; i < quorum; i++)
    {
        cout << COALITION.bestSolution[i] << ",";
    }


    ofstream result_file;
    result_file.open("../data/result/"+std::to_string(puntos)+"_"+std::to_string(puntos_izq)+"_"+std::to_string(puntos_der)+"_"+"_seed_"+mseed+"_result_"+ std::to_string(DATOS.X.size() )+"_"+std::to_string(time(0))+"p.txt");
    result_file << "Time:"<< fixed << time_taken << setprecision(9) << endl;
    result_file << "Minimum Fitness:" << COALITION.bestFitness << endl;
    result_file << "N_BLOCK: " << COALITION.nBlock << " N_THREADS:" << COALITION.nThread << endl;
    result_file << "Coalition:" << endl;
    for (size_t i = 0; i < quorum; i++)
    {
        result_file << COALITION.bestSolution[i] << ",";
    }
    result_file.close();  


    return 0;
}