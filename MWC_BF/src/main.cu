#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <cstdlib>
#include <random>
#include <cstdint>
#include <cstring>
#include <chrono>
#include <vector>


#include <Dataset.cuh>
#include <Coalition.cuh>

using namespace std;


int main() {

    //Dataset DATOS("data/test/Dataset_25_12_13_seed-7.csv");
    //Dataset DATOS("data/test/points_100.txt");
    Dataset DATOS("data/test/points_40.txt");
    //Dataset DATOS("data/votes.json","JSON");
    DATOS.printXY();
    cout << DATOS.getDistanciaHost(0,1) << " - " << DATOS.getDistanciaDevice(0,1) << endl;
    int quorum = trunc(DATOS.X.size() / 2)+1;

    Coalition COALITION(quorum,DATOS.X.size(),DATOS.distMatrix_device,32,1024); 

    // Time variable initialization for execution calculation
    auto initial_time = chrono::high_resolution_clock::now();
    cout << "Generando combinatoria!" << endl;
    COALITION.BestSolution();
    // Stop the clock
    auto final_time = chrono::high_resolution_clock::now();
    double time_taken = chrono::duration_cast<chrono::nanoseconds>(final_time - initial_time).count();

    // Convert the time taken by the algorithm to seconds


    time_taken *= 1e-9;
    cout << "Time:"<< fixed << time_taken << setprecision(9) << endl;
    cout << "Minimum Fitness:" << COALITION.bestFitness << endl;
    cout << "Coalition:" << endl;
    for (size_t i = 0; i < quorum; i++)
    {
        cout << COALITION.bestSolution[i] << ",";
    }


    ofstream result_file;
    result_file.open("../data/result/result_"+ std::to_string(DATOS.X.size() )+".txt");
    result_file << "Time:"<< fixed << time_taken << setprecision(9) << endl;
    result_file << "Minimum Fitness:" << COALITION.bestFitness << endl;
    result_file << "Coalition:" << endl;
    for (size_t i = 0; i < quorum; i++)
    {
        result_file << COALITION.bestSolution[i] << ",";
    }
    result_file.close();  


    return 0;
}