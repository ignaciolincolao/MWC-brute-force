#include "hip/hip_runtime.h"
#include <kernel.cuh>




__device__ float distanciaR2(float x1, float y1, float x2, float y2) {
    return sqrtf((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2));
}

__global__ void computeDistMatrix(float *X, float *Y, float *distMatrix_device, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < n) {
        distMatrix_device[i * n + j] = distanciaR2(X[i], Y[i], X[j], Y[j]);
    }
}


__global__ void evaluate_solution_kernel(int *matrixSolution_device, float *distMatrix_device, float *fitness_device, int nQuorum, int nData, int nSolution){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < nSolution) {
        float sum = 0.0f;
        for (int i = 0; i < nQuorum - 1; ++i) {
            for (int j = i + 1; j < nQuorum; ++j) {
                int pos_i = matrixSolution_device[idx * nQuorum + i];
                int pos_j = matrixSolution_device[idx * nQuorum + j];
                sum += distMatrix_device[pos_i * nData + pos_j];
            }
        }
        fitness_device[idx] = sum;
    }
}

__global__ void evaluate_solution_kernel_v2(int *matrixSolution_device, float *distMatrix_device, float *fitness_device, int nQuorum, int nData, int nSolution){
    extern __shared__ float shared_distMatrix[];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int local_idx = threadIdx.x;

    // Copy distMatrix_device to shared memory
    if (local_idx < nData * nData) {
        shared_distMatrix[local_idx] = distMatrix_device[local_idx];
    }
    __syncthreads();
    // Ensure we don't go out of bounds
    if (idx < nSolution) {
        float sum = 0.0f;
        for (int i = 0; i < nQuorum - 1; ++i) {
            for (int j = i + 1; j < nQuorum; ++j) {
                int pos_i = matrixSolution_device[idx * nQuorum + i];
                int pos_j = matrixSolution_device[idx * nQuorum + j];
                sum += shared_distMatrix[pos_i * nData + pos_j];
            }
        }
        fitness_device[idx] = sum;
    }
}



__global__ void evaluate_solution_kernel_v3(int n, int k, float *distMatrix_device, float *fitness_device, int nSolution, int batchIdx) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x + batchIdx * blockDim.x * gridDim.x;
    int f = idx;
    if (idx >= nSolution) {
        return;  // Ensure we don't go out of bounds
    }

    // Generate the idx-th combination using the Combinadic algorithm
    int a = n;
    int b = k;
    int x = (1 << k) - 1;

    while (idx--) {
        b = k;
        while (x & (1 << (a - 1))) {
            x -= (1 << (a - 1));
            --a;
            --b;
        }
        x |= ((1 << (a - 1)) - 1);
        x &= ~((1 << (a - b - 1)) - 1);
    }

    // Calculate the sum of the pairwise distances for the points in the combination
    float sum = 0.0f;
    for (int i = 0; i < n - 1; ++i) {
        if (!(x & (1 << i))) continue; // Skip if point i is not in the combination
        for (int j = i + 1; j < n; ++j) {
            if (x & (1 << j)) {  // If points i and j are in the combination
                sum += distMatrix_device[i * n + j];
            }
        }
    }

    fitness_device[f] = sum;
}
