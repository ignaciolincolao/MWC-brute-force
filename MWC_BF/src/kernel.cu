#include "hip/hip_runtime.h"
#include <kernel.cuh>




__device__ float distanciaR2(float x1, float y1, float x2, float y2) {
    return sqrtf((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2));
}

__global__ void computeDistMatrix(float *X, float *Y, float *distMatrix_device, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < n) {
        distMatrix_device[i * n + j] = distanciaR2(X[i], Y[i], X[j], Y[j]);
    }
}


__global__ void evaluate_solution_kernel(int *matrixSolution_device, float *distMatrix_device, float *fitness_device, int nQuorum, int nData, int nSolution){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure we don't go out of bounds
    if (idx < nSolution) {
        float sum = 0.0f;
        for (int i = 0; i < nQuorum - 1; ++i) {
            for (int j = i + 1; j < nQuorum; ++j) {
                int pos_i = matrixSolution_device[idx * nQuorum + i];
                int pos_j = matrixSolution_device[idx * nQuorum + j];
                sum += distMatrix_device[pos_i * nData + pos_j];
            }
        }
        fitness_device[idx] = sum;
    }
}
