#include "hip/hip_runtime.h"
#include <kernel.cuh>

__device__ double evaluate_solution_GPU(int* pos, double* mat, int length, int mat_dim) {
    double sum = 0.0;
    for (int i = 0; i <= length - 2; i++) {
        for (int j = i + 1; j <= length - 1; j++) {
            int idx1 = pos[i] * mat_dim + pos[j];
            sum += mat[idx1];
        }
    }
    return sum;
}

__device__ double euclidian_distance_GPU(double x1, double y1, double x2, double y2)
{
	double calculation = pow(pow((x2 - x1), 2) + pow((y2 - y1), 2), 1 / (double)2);
	return calculation;
}

__global__ void calculate_distances(double* new_data, double* distance_matrix, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        for (int j = 0; j < n; j++) {
            distance_matrix[i * n + j] = euclidian_distance_GPU(new_data[i * 2 + 0], new_data[i * 2 + 1], new_data[j * 2 + 0], new_data[j * 2 + 1]);
        }
    }
}