#include <Coalition.cuh>


// Función para comprobar errores de CUDA
inline void checkCudaErrors(hipError_t err, const char* file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "Error en: " << file << ":" << line << " código: "
                  << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}
#define CHECK_CUDA_ERRORS(err) checkCudaErrors(err, __FILE__, __LINE__)

Coalition::Coalition(int nQuorum, int nData,float *distMatrix_device, int nBlock, int nThread):
    distMatrix_device(distMatrix_device), nQuorum(nQuorum),nData(nData),nBlock(nBlock),nThread(nThread){
    nSolution = nBlock*nThread;
    X.resize(nQuorum, 0.0f);
    Y.resize(nQuorum, 0.0f);
    solution.resize(nQuorum, 0);
    bestSolution.resize(nQuorum);
    fitness = DBL_MAX;
    bestFitness = DBL_MAX;
    X_device = nullptr;
    Y_device = nullptr;
    fitness_device = nullptr;
    matrixSolution_device = nullptr;
    matrixSolution_host = (int*) malloc(nQuorum * nSolution * sizeof(int));
    CHECK_CUDA_ERRORS(hipMalloc((void**)&X_device, X.size() * sizeof(float)));
    CHECK_CUDA_ERRORS(hipMalloc((void**)&Y_device, Y.size() * sizeof(float)));
    CHECK_CUDA_ERRORS(hipMalloc((void**)&fitness_device, nSolution * sizeof(float)));
    CHECK_CUDA_ERRORS(hipMalloc((void**)&matrixSolution_device, nQuorum * nSolution * sizeof(int)));
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());
}

Coalition::~Coalition(){
    CHECK_CUDA_ERRORS(hipFree(X_device));
    CHECK_CUDA_ERRORS(hipFree(Y_device));
    CHECK_CUDA_ERRORS(hipFree(fitness_device));
    CHECK_CUDA_ERRORS(hipFree(matrixSolution_device));
    free(matrixSolution_host);
}

void Coalition::BestSolution(){
    vector<int> combination;
    string bitmask(nQuorum, 1); // K leading 1's
    bitmask.resize(nData, 0); // N-K trailing 0's
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device); // get the current device id
    hipGetDeviceProperties(&prop, device);
    int count = 0;
    long long int count_calc = 0;
    do {

        for (int i = 0; i < nData; ++i) { // [0..N-1] integers
            if (bitmask[i]) combination.push_back(i);
        }
        std::copy(combination.begin(), combination.end(), matrixSolution_host + (count % nSolution) * nQuorum);
        combination.clear();
        combination.shrink_to_fit();
        count++;
    
        if (count % nSolution == 0) {
            //auto initial_time = chrono::high_resolution_clock::now();
            CHECK_CUDA_ERRORS(hipMemcpy(matrixSolution_device, matrixSolution_host, nQuorum * nSolution * sizeof(int), hipMemcpyHostToDevice));
            CHECK_CUDA_ERRORS(hipGetLastError());
            CHECK_CUDA_ERRORS(hipDeviceSynchronize());
            int threadsPerBlock = nThread;
            int blocksPerGrid = nBlock;
            //cout << "block: " << blocksPerGrid << " - " << count << endl;
            evaluate_solution_kernel<<<blocksPerGrid, threadsPerBlock>>>(matrixSolution_device, distMatrix_device, fitness_device, nQuorum, nData, nSolution);
            CHECK_CUDA_ERRORS(hipGetLastError());
            CHECK_CUDA_ERRORS(hipDeviceSynchronize());
            find_min_index(nSolution, count, count_calc);
            CHECK_CUDA_ERRORS(hipGetLastError());
            CHECK_CUDA_ERRORS(hipDeviceSynchronize());
            count=0;
            count_calc++;
            //cout << count_calc << endl;
            //auto final_time = chrono::high_resolution_clock::now();
            //double time_taken = chrono::duration_cast<chrono::nanoseconds>(final_time - initial_time).count();
            //time_taken *= 1e-9;
            //cout << "Time:"<< fixed << time_taken << setprecision(9) << count_calc << endl;
            //cout << "Coalition:" << endl;
        }
    } while (prev_permutation(bitmask.begin(), bitmask.end()));

    CHECK_CUDA_ERRORS(hipMemcpy(matrixSolution_device, matrixSolution_host, nQuorum * nSolution * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());
    int threadsPerBlock = nThread;
    int blocksPerGrid = nBlock;
    //cout << "block: " << blocksPerGrid << " - " << count << endl;
    evaluate_solution_kernel<<<blocksPerGrid, threadsPerBlock>>>(matrixSolution_device, distMatrix_device, fitness_device, nQuorum, nData, nSolution);
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());
    find_min_index(nSolution, count,count_calc);
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());
	cout << "Combinaciones:" << count<<endl;
}


void Coalition::find_min_index(int n, int count,int count_calc) {
    thrust::device_ptr<float> fitness_device_ptr(fitness_device);

    thrust::device_ptr<float> min_ptr = thrust::min_element(fitness_device_ptr, fitness_device_ptr + n);

    int min_index = thrust::distance(fitness_device_ptr, min_ptr);
    float min_value = *min_ptr;
    if(min_value < bestFitness){
        bestFitness = min_value;
        CHECK_CUDA_ERRORS(hipMemcpy(&bestSolution[0], matrixSolution_device + min_index * nQuorum, nQuorum * sizeof(int), hipMemcpyDeviceToHost));
        cout << count  << " || " <<  count_calc << " || " << bestFitness << endl;
    }
}


