#include <Dataset.cuh>

// Función para comprobar errores de CUDA
inline void checkCudaErrors(hipError_t err, const char* file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "Error en: " << file << ":" << line << " código: "
                  << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}
#define CHECK_CUDA_ERRORS(err) checkCudaErrors(err, __FILE__, __LINE__)


Dataset::Dataset(const std::string &filename, const std::string type, char separator) {
    if(type == "csv" || type == "CSV"){
        std::ifstream inFile(filename);
        if (!inFile) {
            std::cerr << "No se pudo abrir el archivo " << filename << std::endl;
            exit(EXIT_FAILURE);
        }

        float x, y;
        char actualSeparator;
        while (inFile >> x >> actualSeparator >> y) {
            if (actualSeparator != separator) {
                std::cerr << "El delimitador no coincide con " << separator << std::endl;
                exit(EXIT_FAILURE);
            }
            X.push_back(x);
            Y.push_back(y);
        }
    }
    else{
        if(type == "json" || type == "JSON"){
            ifstream file(filename);
            if (!file) {
                std::cerr << "No se pudo abrir el archivo " << filename << std::endl;
                exit(EXIT_FAILURE);
            }
            json data = json::parse(file);
            cout << "entro" << endl;
            for (size_t i=0; i <data["rollcalls"][0]["votes"].size(); i++){
                X.push_back(data["rollcalls"][0]["votes"][i]["x"]);
                Y.push_back(data["rollcalls"][0]["votes"][i]["y"]);
            }
        }
        else{
            cout << "Tipo de archivo no soportado" << endl;
            exit(EXIT_FAILURE);
        }
    }
    
    int n = X.size();


    CHECK_CUDA_ERRORS(hipMalloc((void**)&X_device, X.size() * sizeof(float)));
    CHECK_CUDA_ERRORS(hipMalloc((void**)&Y_device, Y.size() * sizeof(float)));

    CHECK_CUDA_ERRORS(hipMemcpy(X_device, X.data(), X.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERRORS(hipMemcpy(Y_device, Y.data(), Y.size() * sizeof(float), hipMemcpyHostToDevice));

    CHECK_CUDA_ERRORS(hipMalloc((void**)&distMatrix_device, n * n * sizeof(float)));
    distMatrix_host = (float*) malloc(n * n * sizeof(float));
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());
    generarDistMatrix();
}

Dataset::~Dataset() {
    CHECK_CUDA_ERRORS(hipFree(distMatrix_device));
    CHECK_CUDA_ERRORS(hipFree(X_device));
    CHECK_CUDA_ERRORS(hipFree(Y_device));
    free(distMatrix_host);
}


void Dataset::generarDistMatrix() {
    int n = X.size();
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                    (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    computeDistMatrix<<<numBlocks, threadsPerBlock>>>(X_device, Y_device, distMatrix_device, n);
    
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());
    copyToHost();
}

void Dataset::printXY() const {
    for (size_t i = 0; i < X.size(); ++i) {
        std::cout << "X: " << X[i] << ", Y: " << Y[i] << std::endl;
    }
}


void Dataset::copyToHost() {
    int n = X.size();
    CHECK_CUDA_ERRORS(hipMemcpy(distMatrix_host, distMatrix_device, n * n * sizeof(float), hipMemcpyDeviceToHost));
}

float Dataset::getDistanciaHost(int i, int j) {
    int n = X.size();
    if (i < 0 || i >= n || j < 0 || j >= n) {
        std::cerr << "Índice fuera de rango." << std::endl;
        return -1.0f;
    }
    return distMatrix_host[i * n + j];
}

float Dataset::getDistanciaDevice(int i, int j){
    int n = X.size();
    if (i < 0 || i >= n || j < 0 || j >= n) {
        std::cerr << "Índice fuera de rango." << std::endl;
        return -1.0f;
    }
    float result;
    CHECK_CUDA_ERRORS(hipMemcpy(&result, &distMatrix_device[i * n + j], sizeof(float), hipMemcpyDeviceToHost));
    return result;
}

void Dataset::savePointFile() const{
    ofstream point_file;
    point_file.open("data/save/points_"+std::to_string(X.size())+".txt");
    
    for (size_t i = 0; i < X.size(); i++)
    {
        point_file << X[i] << "," << Y[i] << endl;
    }
    point_file.close();    
}